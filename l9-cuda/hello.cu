 

#include <hip/hip_runtime.h>
#include <stdio.h>
 
//const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(int *a, int *b) 
{
	int z = blockDim.x * blockIdx.x + threadIdx.x;
	a[z] = b[z]* b[z] * b[z];
	//a[threadIdx.x] = 5;
}
 
int main(int argc, char** argv)
{
	int SIZE = atoi(argv[1]);
	int *a =(int*) malloc(SIZE * sizeof(int));
	int *b =(int*) malloc(SIZE * sizeof(int));

	for(int i=0; i<SIZE; i++) {
		a[i]=0;
		b[i]=rand()%100+1;
	}
//	int a[N] = {};
	//int b[N] = {16, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	
 
	int *ad;
	int *bd;
	const int csize = SIZE*sizeof(int);
	const int isize = SIZE*sizeof(int);
 
	printf("input ");
	for (int i=0;i<SIZE;i++)
		printf("%d ", b[i]);
	printf("\n");

	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	//hello<<<dimGrid, dimBlock>>>(ad, bd);
	int cores = 512;
	//int blocks = SIZE/cores + 16;
	int blocks = 16;
	int threadsPerBlock = 256;
	//int blocksPerGrid =(SIZE + threadsPerBlock – 1) / threadsPerBlock;

	int blocksPerGrid = ( SIZE + threadsPerBlock -1) / threadsPerBlock;
	printf("blocks = %d cores = %d\n", blocks, cores);
	hello<<<blocksPerGrid, threadsPerBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("ouput ");
	for (int i=0;i<SIZE;i++)
		printf("%d ", a[i]);
	printf("\n");
	//printf("%s\n", a);
	free(a);
	free(b);
	return EXIT_SUCCESS;
}
