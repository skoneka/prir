// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 

#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(int *a, int *b) 
{
	a[threadIdx.x] = b[threadIdx.x]*b[threadIdx.x]*b[threadIdx.x];
	//a[threadIdx.x] = 5;
}
 
int main()
{
	int a[N] = {};
	int b[N] = {16, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	int *ad;
	int *bd;
	const int csize = N*sizeof(int);
	const int isize = N*sizeof(int);
 
	printf("input ");
	for (int i=0;i<N;i++)
		printf("%d ", b[i]);
	printf("\n");

	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	//hello<<<dimGrid, dimBlock>>>(ad, bd);
	hello<<<1, 16>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("ouput ");
	for (int i=0;i<N;i++)
		printf("%d ", a[i]);
	printf("\n");
	//printf("%s\n", a);
	return EXIT_SUCCESS;
}
